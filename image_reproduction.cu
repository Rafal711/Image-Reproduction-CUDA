#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include ""

#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>

#include <iostream>
#include <string>
#include <vector>

#include <stdio.h>
#include <math.h>

#define N 1024 // max number of threads in one block

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

///////////////////////////////////////////////////////////////////////////////////////
// CROSSOVER

enum Crossover_method {
    SINGLE_POINT,
    TWO_POINT,
    UNIFORM
};


// crossover operation functions
// running as CHILD kernels - 1D grid, block -> crossover of one pair, thread in a block -> gene in a chromosome 

// single single-point crossover operation
__global__ void single_point(const uint8_t* parents[], uint8_t* offsprings[], const int* crossover_point) {

    int gene_idx = threadIdx.x;   // index of a gene in a parent chromosome
    int pair_idx = 2 * blockIdx.x;    // index of a pair of parents

    int offspring_idx = pair_idx;

    if (gene_idx <= *crossover_point) {
        offsprings[offspring_idx][gene_idx] = parents[pair_idx][gene_idx];
        offsprings[offspring_idx+1][gene_idx] = parents[pair_idx+1][gene_idx];
    }
    else {
        offsprings[offspring_idx][gene_idx] = parents[pair_idx+1][gene_idx];
        offsprings[offspring_idx+1][gene_idx] = parents[pair_idx][gene_idx];
    }

}


// single two-point crossover operation
__global__ void two_point(const uint8_t* parents[], uint8_t* offsprings[],
                          const int* first_crossover_point, const int* second_crossover_point) {

    int gene_idx = threadIdx.x;   // index of a gene in a parent chromosome
    int pair_idx = 2 * blockIdx.x;    // index of a pair of parents

    int offspring_idx = pair_idx;

    if (gene_idx <= *first_crossover_point || gene_idx > *second_crossover_point) {
        offsprings[offspring_idx][gene_idx] = parents[pair_idx][gene_idx];
        offsprings[offspring_idx + 1][gene_idx] = parents[pair_idx+1][gene_idx];
    }
    else {
        offsprings[offspring_idx][gene_idx] = parents[pair_idx+1][gene_idx];
        offsprings[offspring_idx + 1][gene_idx] = parents[pair_idx][gene_idx];
    }
}


// single uniform crossover operation
__global__ void uniform(const uint8_t* parents[], uint8_t* offsprings[], const bool* crossover_mask) {

    int gene_idx = threadIdx.x;   // index of a gene in a parent chromosome
    int pair_idx = 2* blockIdx.x;    // index of a pair of parents
    int offspring_idx = pair_idx;

    if (crossover_mask[gene_idx]) {
        offsprings[offspring_idx][gene_idx] = parents[pair_idx+1][gene_idx];
        offsprings[offspring_idx + 1][gene_idx] = parents[pair_idx][gene_idx];
    }
    else {
        offsprings[offspring_idx][gene_idx] = parents[pair_idx][gene_idx];
        offsprings[offspring_idx + 1][gene_idx] = parents[pair_idx+1][gene_idx];
    }
}


__device__ void generate_random_unique_array(hiprandState* state, const int state_idx, int* tab, const int length, const int max) {
    for (int i = 0; i < length; i++) {
        bool unique = 0;
        while (!unique) {
            unique = 1;
            float random = hiprand_uniform(&state[state_idx]);
            random *= (max + 0.999999);
            tab[i] = (int)truncf(random);
            for (int j = 0; j < i; j++) {
                if (tab[i] == tab[j]) {
                    unique = 0;
                    break;
                }
            }
        }
    } 
}


// crossover over a given mating pool (called for every thread (algorithm) in reproduce_image())
__device__ void perform_crossover(const uint8_t** mating_pool[], uint8_t** offsprings[],
                                  const Crossover_method method, const int* combinations,
                                  const int mating_pool_length, const int number_of_offsprings, 
                                  hiprandState* state) {

    int algorithm_idx = threadIdx.x;
    const int number_of_crossovers = number_of_offsprings / 2;

    // allocating memory for a 2D array of parent chromosomes (every 2 make a pair)
    const uint8_t** parents = new uint8_t*[number_of_crossovers*2];

    // generate array of random integers (pair idx in combinations array), no repetitions
    int* rand_int = new int[number_of_crossovers];
    generate_random_unique_array(state, algorithm_idx, rand_int, number_of_crossovers, mating_pool_length*(mating_pool_length-1)/2);

    // pick pairs based on randomly generated array
    for (int i = 0; i < number_of_crossovers; i++) {
        parents[2 * i] = mating_pool[algorithm_idx][combinations[2 * rand_int[i]]];
        parents[2 * i + 1] = mating_pool[algorithm_idx][combinations[2 * rand_int[i] + 1]];
    }
    
    // launching crossover as child kernels
    float random;
    if (method == Crossover_method::SINGLE_POINT) {
        random = hiprand_uniform(&state[algorithm_idx]) * (1024 + 0.999999);
        int crossover_point = (int)truncf(random);
        single_point << <number_of_crossovers, 1024 >> > (parents, offsprings[algorithm_idx], &crossover_point);
    }
    else if (method == Crossover_method::TWO_POINT) {
        random = hiprand_uniform(&state[algorithm_idx]) * (1024 + 0.999999);
        int first_crossover_point = (int)truncf(random);
        random = hiprand_uniform(&state[algorithm_idx]) * (1024 + 0.999999);
        int second_crossover_point = (int)truncf(random);
        two_point << <number_of_crossovers, 1024 >> > (parents, offsprings[algorithm_idx], &first_crossover_point, &second_crossover_point);
    }
    else {
        bool crossover_mask[1024];
        //generate nr of genes to crossover
        for (int i = 0; i < 1024; i++) {
            random = hiprand_uniform(&state[algorithm_idx]) * (1 + 0.999999);
            crossover_mask[i] = (int)truncf(random);
        }
        uniform << <number_of_crossovers, 1024>> > (parents, offsprings[algorithm_idx], crossover_mask);
    }
    
    // freeing the memory
    delete[] parents;
    delete[] rand_int;
}

///////////////////////////////////////////////////////////////////////////////////////


// main loop of the algorithm (kernel - 1 block with N threads, where 1 thread is 1 algorithm)
__global__ void reproduce_image(uint8_t base_chromosome) {


}


__global__ void setup_curand(hiprandState* state) {
    int idx = threadIdx.x;
    hiprand_init(1234, idx, 0, &state[idx]);
}


void cvMatToGrayArray(const cv::Mat& image, uint8_t* array) {
    cv::Mat greyMat;
    cv::cvtColor(image, greyMat, cv::COLOR_BGR2GRAY);

    for (int i = 0; i < greyMat.rows * greyMat.cols; i++) {
        array[i] = *(greyMat.begin<uint8_t>() + i);
    }
}

cv::Mat grayArrayToCvMat(const cv::Mat& image, const uint8_t* array) {
    cv::Mat reshaped = cv::Mat(image.rows * image.cols, 1, CV_8UC1, (unsigned*)array).reshape(0, image.rows);
    return reshaped;
}

__host__ void generate_all_idx_combinations(int* pairs, int max_idx) {

    int counter = 0;
    for (int i = 0; i < max_idx; i++) {
        for (int j = i; j < max_idx; j++) {
            pairs[2 * counter] = i;
            pairs[2 * counter + 1] = j;
        }
    }
}

__global__ void setup_kernel_multi_blocks(hiprandState* state, uint16_t seed_offset)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed_offset + idx, idx, 0, &state[idx]);
}

__global__ void populationInit_multi_blocks(uint8_t* population, hiprandState* state) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    const int range{ 255 };
    population[idx] = (uint8_t)(ceil((hiprand_uniform(&(state[idx])) * (range + 1))) - 1);;
}

__host__ void run_program() {
    std::string image_path = "tangerines.jpg";
    cv::Mat image = cv::imread(image_path);
    uint8_t* grayArray = new uint8_t[image.rows * image.cols];
    cvMatToGrayArray(image, grayArray);

    uint16_t number_of_iterations = 10000;

    // GA reproduction parameters
    uint16_t number_of_parents = 4;
    uint16_t number_of_offsprings = 8;

    // mutation parameters
    float mutation_percentage = 0.01;

    // termination condition
    float epsilon = 1.0E-12F;
    uint16_t terminate_after = 500;

    uint16_t nr_of_parallel_algorithms = 3;
    

    uint16_t chromosome_size = 5;
    uint16_t population_size = 4;
    uint16_t number_of_threads = chromosome_size;
    uint16_t number_of_blocks = population_size * nr_of_parallel_algorithms;
    uint16_t multiple_population_size = population_size * nr_of_parallel_algorithms * chromosome_size;


    // HOST concatenated population allocation
    uint8_t* h_mpopulation = new uint8_t[multiple_population_size];

    // DEVICE concatenated population allocation
    uint8_t* d_mpopulation;
    CUDA_CALL(hipMalloc(&d_mpopulation, sizeof(uint8_t) * multiple_population_size));

    // DEVICE hiprandState initialization
    hiprandState* devStates;
    CUDA_CALL(hipMalloc((void**)&devStates, multiple_population_size * sizeof(hiprandState)));

    // ------------------------------------------ genetic algorithm start ------------------------------------------
    setup_kernel_multi_blocks <<< number_of_blocks, number_of_threads >>> (devStates, 7);
    populationInit_multi_blocks <<< number_of_blocks, number_of_threads >>> (d_mpopulation, devStates);








    // ------------------------------------------ genetic algorithm end ------------------------------------------

    CUDA_CALL(hipMemcpy(h_mpopulation, d_mpopulation, sizeof(uint8_t) * multiple_population_size, hipMemcpyDeviceToHost));
    hipDeviceSynchronize();

    // show results
    for (std::size_t i = 0; i < multiple_population_size; ++i) {
        std::cout << static_cast<int>(h_mpopulation[i]) << " ";
    }

    // free all device memory
    hipFree(d_mpopulation);
    // free all host memory
    delete[] h_mpopulation;







    // for random number generation on GPU
    hiprandState* d_state;
    std::cout << d_state->d << std::endl;
    hipMalloc(&d_state, sizeof(hiprandState));
    setup_curand << <1, nr_of_parallel_algorithms >> > (d_state);

    
    //hipDeviceSynchronize();
    size_t combinations_nr = number_of_parents * (number_of_parents - 1) / 2;
    int* combinations = new int[combinations_nr];
    generate_all_idx_combinations(combinations, number_of_parents - 1);
    hipMalloc(&combinations, combinations_nr * sizeof(int));
    //perform_crossover << < >> > ();



    // imgRGB2chromosome(...)
    
    // kernel launch
    // reproduce_image(...);

    delete[] combinations;
    hipFree(&d_state);
    ////Display the result image
    //cv::Mat resultImage = grayArrayToCvMat(image, grayArray);
    //cv::imwrite("Result.jpg", image);
    //cv::namedWindow("Result image", cv::WINDOW_AUTOSIZE);
    //cv::imshow("Result image", resultImage);
    //cv::moveWindow("Result image", 0, 45);
    //cv::waitKey(0);
    //cv::destroyAllWindows();
 
}


int main()
{
    run_program();
    return 0;
}
